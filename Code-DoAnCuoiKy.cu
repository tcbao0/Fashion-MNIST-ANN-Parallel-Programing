
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>

#define IMAGE_ROWS 28
#define IMAGE_COLS 28
#define INPUT_SIZE 784
#define HIDDEN_SIZE_1 128
#define HIDDEN_SIZE_2 128
#define OUTPUT_SIZE 10
#define EPOCHS 3
#define LEARNING_RATE 0.01

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(EXIT_FAILURE);\
    }\
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

uint32_t swapEndian(uint32_t test) {
    return ((test & 0xFF) << 24) | 
           ((test & 0xFF00) << 8) | 
           ((test & 0xFF0000) >> 8) | 
           ((test & 0xFF000000) >> 24);
}

unsigned char** readImages(const char* fileName, int* numImages, int* numRows, int* numCols) {
    FILE* f = fopen(fileName, "rb");
	if (f == NULL)
    {
        printf("Failed to open the IDX3-UBYTE file: %s\n", fileName);
        return NULL;
    }

    uint32_t magicNumber, nImages, nRows, nCols;

    fread(&magicNumber, sizeof(uint32_t), 1, f);
    fread(&nImages, sizeof(uint32_t), 1, f);
    fread(&nRows, sizeof(uint32_t), 1, f);
    fread(&nCols, sizeof(uint32_t), 1, f);

    // Chuyển đổi thứ tự byte
    magicNumber = swapEndian(magicNumber);
    nImages = swapEndian(nImages);
    nRows = swapEndian(nRows);
    nCols = swapEndian(nCols);

    *numImages = nImages;
    *numRows = nRows;
    *numCols = nCols;

    unsigned char** images = (unsigned char**)malloc(nImages * sizeof(unsigned char*));
    for (int i = 0; i < nImages; i++) {
        images[i] = (unsigned char*)malloc(nRows * nCols * sizeof(unsigned char));
        fread(images[i], sizeof(unsigned char), nRows * nCols, f);
    }

    fclose(f);
    return images;
}

unsigned char* readLabels(const char* fileName, int* numLabels) {
    FILE* f = fopen(fileName, "rb");
	if (f == NULL)
    {
        printf("Failed to open the IDX1-UBYTE file: %s\n", fileName);
        return NULL;
    }

    uint32_t magicNumber, nLabels;

    fread(&magicNumber, sizeof(uint32_t), 1, f);
    fread(&nLabels, sizeof(uint32_t), 1, f);

    // Chuyển đổi thứ tự byte
    magicNumber = swapEndian(magicNumber);
    nLabels = swapEndian(nLabels);

    *numLabels = nLabels;

    unsigned char* labels = (unsigned char*)malloc(nLabels * sizeof(unsigned char));
    fread(labels, sizeof(unsigned char), nLabels, f);

    fclose(f);
    return labels;
}

float sigmoid(float x) {
    return 1.0f / (1.0f + expf(-x));
}

float sigmoid_derivative(float x) {
    return x * (1.0f - x);
}

// Hàm softmax cho lớp output
void softmax(float* x, int size) {
    float sum = 0.0f;
    for (int i = 0; i < size; i++) {
        x[i] = expf(x[i]);
        sum += x[i];
    }
    for (int i = 0; i < size; i++) {
        x[i] /= sum;
    }
}

void forwardLayerCPU(float* inputLayer, float* weights, float* biases, float* outputLayer, int inputSize, int outputSize, bool applySigmoid = true) {
    for (int j = 0; j < outputSize; j++) {
        outputLayer[j] = biases[j];
        for (int k = 0; k < inputSize; k++) {
            outputLayer[j] += inputLayer[k] * weights[k * outputSize + j];
        }
        if (applySigmoid) {
            outputLayer[j] = sigmoid(outputLayer[j]);
        }
    }
}

__global__ void forwardLayerGPU(float* inputLayer, float* weights, float* biases, float* outputLayer, int inputSize, int outputSize, bool applySigmoid) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < outputSize) {
        float sum = biases[idx];
        for (int i = 0; i < inputSize; i++) {
            sum += inputLayer[i] * weights[i * outputSize + idx];
        }
        outputLayer[idx] = applySigmoid ? 1.0f / (1.0f + expf(-sum)) : sum;
    }
}


void calculateCValue(float* outputDelta, float* outputLayer, unsigned char* trainLabels, int outputSize, int i) {
    for (int j = 0; j < outputSize; j++) {
        outputDelta[j] = (trainLabels[i] == j ? 1.0f : 0.0f) - outputLayer[j];
    }
}



void calculateDeltaLayer(float* currentLayer, float* nextLayerDelta, 
                         float* currentLayerDelta, float* weights, 
                         int currentLayerSize, int nextLayerSize) {
    
    // Tính delta cho lớp hiện tại từ delta của lớp tiếp theo
    for (int j = 0; j < currentLayerSize; j++) {
        currentLayerDelta[j] = 0;
        for (int k = 0; k < nextLayerSize; k++) {
            currentLayerDelta[j] += nextLayerDelta[k] * weights[j * nextLayerSize + k];
        }
        currentLayerDelta[j] *= sigmoid_derivative(currentLayer[j]);
    }
}

__global__ void computeDeltaGPU(float* currentLayer, float* nextLayerDelta, float* currentLayerDelta, float* weights, int currentSize, int nextSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < currentSize) {
        float sum = 0;
        for (int j = 0; j < nextSize; j++) {
            sum += nextLayerDelta[j] * weights[idx * nextSize + j];
        }
        currentLayerDelta[idx] = sum * currentLayer[idx] * (1.0f - currentLayer[idx]);
    }
}

void updateWeights(float* weights, float* biases, float* layer, float* delta, 
                   int layerSize, int prevLayerSize, float learningRate) {
    // Cập nhật trọng số
    for (int j = 0; j < layerSize; j++) {
        for (int k = 0; k < prevLayerSize; k++) {
            weights[k * layerSize + j] += learningRate * layer[k] * delta[j];
        }
    }
    
    // Cập nhật bias
    for (int j = 0; j < layerSize; j++) {
        biases[j] += learningRate * delta[j];
    }
}

__global__ void updateWeightsGPU(float* weights, float* biases, float* layer, float* delta, int layerSize, int prevLayerSize, float learningRate) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < layerSize) {
        for (int j = 0; j < prevLayerSize; j++) {
            atomicAdd(&weights[j * layerSize + idx], learningRate * layer[j] * delta[idx]);
        }
        atomicAdd(&biases[idx], learningRate * delta[idx]);
    }
}

float computeFinalAccuracy(unsigned char** testImages, unsigned char* testLabels, int numTestImages, int numRows, int numCols, float* hiddenWeights1, float* hiddenWeights2, float* outputWeights, float* hiddenBiases1, float* hiddenBiases2, float* outputBiases) {
    int correct = 0;
    for (int i = 0; i < numTestImages; i++) {
        float inputLayer[INPUT_SIZE];
        for (int j = 0; j < INPUT_SIZE; j++) {
            inputLayer[j] = testImages[i][j] / 255.0f; // Chuẩn hóa giá trị đầu vào
        }

        float* hiddenLayer1 = (float*)malloc(HIDDEN_SIZE_1 * sizeof(float));
        forwardLayerCPU(inputLayer, hiddenWeights1, hiddenBiases1, hiddenLayer1, INPUT_SIZE, HIDDEN_SIZE_1);

        float* hiddenLayer2 = (float*)malloc(HIDDEN_SIZE_2 * sizeof(float));
        forwardLayerCPU(hiddenLayer1, hiddenWeights2, hiddenBiases2, hiddenLayer2, HIDDEN_SIZE_1, HIDDEN_SIZE_2);

        float* outputLayer = (float*)malloc(OUTPUT_SIZE * sizeof(float));
        forwardLayerCPU(hiddenLayer2, outputWeights, outputBiases, outputLayer, HIDDEN_SIZE_2, OUTPUT_SIZE, false);

        // Áp dụng softmax để có xác suất cho mỗi lớp
        softmax(outputLayer, OUTPUT_SIZE);

        // Kiểm tra kết quả dự đoán
        int predictedLabel = 0;
        for (int j = 1; j < OUTPUT_SIZE; j++) {
            if (outputLayer[j] > outputLayer[predictedLabel]) {
                predictedLabel = j;
            }
        }

        if (predictedLabel == testLabels[i]) {
            correct++;
        }
    }
    return (float)correct / numTestImages;
}

__global__ void normalizeInputGPU(float* input, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        input[idx] = input[idx] / 255.0f; // Normalize to range [0, 1]
    }
}

__global__ void softmaxGPU(float* output, int size)
{
    __shared__ float max_val;
    __shared__ float sum;

    if (threadIdx.x == 0) {
        max_val = -INFINITY;
        sum = 0.0f;
    }
    __syncthreads();

    float local_max = -INFINITY;
    for (int i = threadIdx.x; i < size; i += blockDim.x) 
    {
        local_max = fmaxf(local_max, output[i]);
    }
    atomicAdd(&max_val, local_max);
    __syncthreads();

    float local_sum = 0;
    for (int i = threadIdx.x; i < size; i += blockDim.x)
    {
        float exp_val = expf(output[i] - max_val);
        output[i] = exp_val;
        local_sum += exp_val;
    }
    atomicAdd(&sum, local_sum);
    __syncthreads();

    for (int i = threadIdx.x; i < size; i += blockDim.x)
    {
        output[i] /= sum;
    }
}


__global__ void calculateOutputDeltaGPU(float* outputDelta, float* outputLayer, int* labels, int outputSize, int batchSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < outputSize * batchSize) {
        int sample = idx / outputSize;
        int label = idx % outputSize;
        outputDelta[idx] = (labels[sample] == label ? 1.0f : 0.0f) - outputLayer[idx];
    }
}

__global__ void computeAccuracyKernel(float* hiddenWeights1, float* hiddenWeights2, float* outputWeights,
                                      float* hiddenBiases1, float* hiddenBiases2, float* outputBiases,
                                      float* testImages, unsigned char* testLabels, int numTestImages,
                                      int* correctPredictions) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numTestImages) {
        float inputLayer[INPUT_SIZE];
        for (int j = 0; j < INPUT_SIZE; j++) {
            inputLayer[j] = testImages[idx * INPUT_SIZE + j] / 255.0f;
        }

        float hiddenLayer1[HIDDEN_SIZE_1];
        float hiddenLayer2[HIDDEN_SIZE_2];
        float outputLayer[OUTPUT_SIZE];

        // Forward pass
        forwardLayerGPU<<<1, HIDDEN_SIZE_1>>>(inputLayer, hiddenWeights1, hiddenBiases1, hiddenLayer1, INPUT_SIZE, HIDDEN_SIZE_1, true);
        forwardLayerGPU<<<1, HIDDEN_SIZE_2>>>(hiddenLayer1, hiddenWeights2, hiddenBiases2, hiddenLayer2, HIDDEN_SIZE_1, HIDDEN_SIZE_2, true);
        forwardLayerGPU<<<1, OUTPUT_SIZE>>>(hiddenLayer2, outputWeights, outputBiases, outputLayer, HIDDEN_SIZE_2, OUTPUT_SIZE, false);
        softmaxGPU<<<1, OUTPUT_SIZE>>>(outputLayer, OUTPUT_SIZE);

        // Find predicted label
        int predictedLabel = 0;
        for (int j = 1; j < OUTPUT_SIZE; j++) {
            if (outputLayer[j] > outputLayer[predictedLabel]) {
                predictedLabel = j;
            }
        }

        if (predictedLabel == testLabels[idx]) {
            atomicAdd(correctPredictions, 1);
        }
    }
}

float computeAccuracyGPU(float* d_hiddenWeights1, float* d_hiddenWeights2, float* d_outputWeights,
    float* d_hiddenBiases1, float* d_hiddenBiases2, float* d_outputBiases,
    unsigned char** testImages, unsigned char* testLabels, int numTestImages) {
float* d_testImages;
unsigned char* d_testLabels;
int* d_correctPredictions;
int h_correctPredictions = 0;

CHECK(hipMalloc(&d_testImages, numTestImages * INPUT_SIZE * sizeof(float)));
CHECK(hipMalloc(&d_testLabels, numTestImages * sizeof(unsigned char)));
CHECK(hipMalloc(&d_correctPredictions, sizeof(int)));

// Copy test data to device
for (int i = 0; i < numTestImages; i++) {
CHECK(hipMemcpy(d_testImages + i * INPUT_SIZE, testImages[i], INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
}
CHECK(hipMemcpy(d_testLabels, testLabels, numTestImages * sizeof(unsigned char), hipMemcpyHostToDevice));
CHECK(hipMemset(d_correctPredictions, 0, sizeof(int)));

dim3 blockDim(256);
dim3 gridDim((numTestImages + blockDim.x - 1) / blockDim.x);

computeAccuracyKernel<<<gridDim, blockDim>>>(d_hiddenWeights1, d_hiddenWeights2, d_outputWeights,
                            d_hiddenBiases1, d_hiddenBiases2, d_outputBiases,
                            d_testImages, d_testLabels, numTestImages,
                            d_correctPredictions);

CHECK(hipMemcpy(&h_correctPredictions, d_correctPredictions, sizeof(int), hipMemcpyDeviceToHost));

hipFree(d_testImages);
hipFree(d_testLabels);
hipFree(d_correctPredictions);

return (float)h_correctPredictions / numTestImages;
}


int training(unsigned char** trainImages, unsigned char* trainLabels, unsigned char** testImages, unsigned char* testLabels, int numTrainImages, int numTestImages, int numRows, int numCols, int flag) {
    GpuTimer timer;

    // Thời gian xử lý truyền qua từng lớp
    float timeInputLayer = 0, timeHiddenLayer1 = 0, timeHiddenLayer2 = 0, timeOutputLayer = 0;
    // Thời gian cập nhật trọng số mô hình
    float timeInputHidden1 = 0, timeHidden1Hidden2 = 0, timeHidden2Output = 0;
    float finalAccuracy = 0;

    if (flag == 1) {
        // Cấp phát bộ nhớ cho trọng số của các layers
        float* hiddenWeights1 = (float*)malloc(INPUT_SIZE * HIDDEN_SIZE_1 * sizeof(float));
        float* hiddenWeights2 = (float*)malloc(HIDDEN_SIZE_1 * HIDDEN_SIZE_2 * sizeof(float));
        float* outputWeights = (float*)malloc(HIDDEN_SIZE_2 * OUTPUT_SIZE * sizeof(float));
        float* hiddenBiases1 = (float*)malloc(HIDDEN_SIZE_1 * sizeof(float));
        float* hiddenBiases2 = (float*)malloc(HIDDEN_SIZE_2 * sizeof(float));
        float* outputBiases = (float*)malloc(OUTPUT_SIZE * sizeof(float));

        // Thực hiện khởi tạo ngẫu nhiên trọng số ban đầu
        for (int i = 0; i < INPUT_SIZE * HIDDEN_SIZE_1; i++) hiddenWeights1[i] = ((float)rand() / RAND_MAX - 0.5f) * 2;
        for (int i = 0; i < HIDDEN_SIZE_1 * HIDDEN_SIZE_2; i++) hiddenWeights2[i] = ((float)rand() / RAND_MAX - 0.5f) * 2;
        for (int i = 0; i < HIDDEN_SIZE_2 * OUTPUT_SIZE; i++) outputWeights[i] = ((float)rand() / RAND_MAX - 0.5f) * 2;
        // Gán biases bắt đầu là 0
        for (int i = 0; i < HIDDEN_SIZE_1; i++) hiddenBiases1[i] = 0;
        for (int i = 0; i < HIDDEN_SIZE_2; i++) hiddenBiases2[i] = 0;
        for (int i = 0; i < OUTPUT_SIZE; i++) outputBiases[i] = 0;

        // Duyệt qua từng epoch và huấn luyện mô hình
        for (int epoch = 1; epoch <= EPOCHS; epoch++) {
            for (int i = 0; i < numTrainImages; i++) {
                // Quá trình feedforward
                // Xử lý lớp Input và đo thời gian
                timer.Start();
                float* inputLayer = (float*)malloc(INPUT_SIZE * sizeof(float));
                for (int j = 0; j < INPUT_SIZE; j++) {
                    inputLayer[j] = trainImages[i][j] / 255.0f; // Normalize
                }
                timer.Stop();
                timeInputLayer += timer.Elapsed();
                // Xử lý lớp Hidden 1 và đo thời gian
                timer.Start();
                float* hiddenLayer1 = (float*)malloc(HIDDEN_SIZE_1 * sizeof(float));
                forwardLayerCPU(inputLayer, hiddenWeights1, hiddenBiases1, hiddenLayer1, INPUT_SIZE, HIDDEN_SIZE_1);
                timer.Stop();
                timeHiddenLayer1 += timer.Elapsed();
                // Xử lý lớp Hidden 2 và đo thời gian
                timer.Start();
                float* hiddenLayer2 = (float*)malloc(HIDDEN_SIZE_2 * sizeof(float));
                forwardLayerCPU(hiddenLayer1, hiddenWeights2, hiddenBiases2, hiddenLayer2, HIDDEN_SIZE_1, HIDDEN_SIZE_2);
                timer.Stop();
                timeHiddenLayer2 += timer.Elapsed();
                // Xử lý lớp Output và đo thời gian
                timer.Start();
                float* outputLayer = (float*)malloc(OUTPUT_SIZE * sizeof(float));
                forwardLayerCPU(hiddenLayer2, outputWeights, outputBiases, outputLayer, HIDDEN_SIZE_2, OUTPUT_SIZE, false);
                timer.Stop();
                timeOutputLayer += timer.Elapsed();
                // Áp dụng softmax để có xác suất cho mỗi lớp
                softmax(outputLayer, OUTPUT_SIZE);
                // Quá trình backpropagation
                // Backpropagation để cập nhật trọng số và bias
                float* outputDelta = (float*)malloc(OUTPUT_SIZE * sizeof(float));
                calculateCValue(outputDelta, outputLayer, trainLabels, OUTPUT_SIZE, i);
                // Tính gardient và cập nhật trọng số cho lớp output
                timer.Start();
                updateWeights(outputWeights, outputBiases, hiddenLayer2, outputDelta, OUTPUT_SIZE, HIDDEN_SIZE_2, LEARNING_RATE);
                timer.Stop();
                timeHidden2Output += timer.Elapsed();
                // Tính gardient và cập nhật trọng số cho lớp ẩn 2
                timer.Start();
                float* hiddenLayer2Delta = (float*)malloc(HIDDEN_SIZE_2 * sizeof(float));
                calculateDeltaLayer(hiddenLayer2, outputDelta, hiddenLayer2Delta, outputWeights, HIDDEN_SIZE_2, OUTPUT_SIZE);
                updateWeights(hiddenWeights2, hiddenBiases2, hiddenLayer1, hiddenLayer2Delta, HIDDEN_SIZE_2, HIDDEN_SIZE_1, LEARNING_RATE);
                timer.Stop();
                timeHidden1Hidden2 += timer.Elapsed();
                // Tính gardient và cập nhật trọng số cho lớp ẩn 1
                timer.Start();
                float* hiddenLayer1Delta = (float*)malloc(HIDDEN_SIZE_1 * sizeof(float));
                calculateDeltaLayer(hiddenLayer1, hiddenLayer2Delta, hiddenLayer1Delta, hiddenWeights2, HIDDEN_SIZE_1, HIDDEN_SIZE_2);
                updateWeights(hiddenWeights1, hiddenBiases1, inputLayer, hiddenLayer1Delta, HIDDEN_SIZE_1, INPUT_SIZE, LEARNING_RATE);
                timer.Stop();
                timeInputHidden1 += timer.Elapsed();
                free(inputLayer);
                free(hiddenLayer1);
                free(hiddenLayer2);
                free(outputLayer);
                free(outputDelta);
                free(hiddenLayer2Delta);
                free(hiddenLayer1Delta);
            }

            printf("Epoch %d: Accuracy = %.4f\n", epoch, computeFinalAccuracy(testImages, testLabels, numTestImages, numRows, numCols, hiddenWeights1, hiddenWeights2, outputWeights, hiddenBiases1, hiddenBiases2, outputBiases));
        }
        finalAccuracy = computeFinalAccuracy(testImages, testLabels, numTestImages, numRows, numCols, hiddenWeights1, hiddenWeights2, outputWeights, hiddenBiases1, hiddenBiases2, outputBiases);

        free(hiddenWeights1);
        free(hiddenWeights2);
        free(outputWeights);
        free(hiddenBiases1); 
        free(hiddenBiases2);
        free(outputBiases);
    }
    else if (flag == 2) {
        float *d_inputLayer, *d_hiddenWeights1, *d_hiddenWeights2, *d_outputWeights;
        float *d_hiddenBiases1, *d_hiddenBiases2, *d_outputBiases;
        float *d_hiddenLayer1, *d_hiddenLayer2, *d_outputLayer;
        float *d_outputDelta, *d_hiddenLayer2Delta, *d_hiddenLayer1Delta;
        int *d_trainLabels;


        CHECK(hipMalloc(&d_inputLayer, INPUT_SIZE * sizeof(float)));
        CHECK(hipMalloc(&d_hiddenWeights1, INPUT_SIZE * HIDDEN_SIZE_1 * sizeof(float)));
        CHECK(hipMalloc(&d_hiddenWeights2, HIDDEN_SIZE_1 * HIDDEN_SIZE_2 * sizeof(float)));
        CHECK(hipMalloc(&d_outputWeights, HIDDEN_SIZE_2 * OUTPUT_SIZE * sizeof(float)));
        CHECK(hipMalloc(&d_hiddenBiases1, HIDDEN_SIZE_1 * sizeof(float)));
        CHECK(hipMalloc(&d_hiddenBiases2, HIDDEN_SIZE_2 * sizeof(float)));
        CHECK(hipMalloc(&d_outputBiases, OUTPUT_SIZE * sizeof(float)));
        CHECK(hipMalloc(&d_hiddenLayer1, HIDDEN_SIZE_1 * sizeof(float)));
        CHECK(hipMalloc(&d_hiddenLayer2, HIDDEN_SIZE_2 * sizeof(float)));
        CHECK(hipMalloc(&d_outputLayer, OUTPUT_SIZE * sizeof(float)));
        CHECK(hipMalloc(&d_outputDelta, OUTPUT_SIZE * sizeof(float)));
        CHECK(hipMalloc(&d_hiddenLayer2Delta, HIDDEN_SIZE_2 * sizeof(float)));
        CHECK(hipMalloc(&d_hiddenLayer1Delta, HIDDEN_SIZE_1 * sizeof(float)));
        CHECK(hipMalloc(&d_trainLabels, numTrainImages * sizeof(int)));

        CHECK(hipMemcpy(d_trainLabels, trainLabels, numTrainImages * sizeof(int), hipMemcpyHostToDevice));

        dim3 blockDim(128);

        for (int epoch = 0; epoch <= EPOCHS; epoch++)
        {   
            //TODO
            for (int i = 0; i < numTrainImages; i++)
            {
                CHECK(hipMemcpy(d_inputLayer, trainImages[i], INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
                
                // Normalize input
                timer.Start();
                dim3 inputGridDim((INPUT_SIZE + blockDim.x - 1) / blockDim.x);
                normalizeInputGPU<<<inputGridDim, blockDim>>>(d_inputLayer, INPUT_SIZE);
                hipDeviceSynchronize();
                timer.Stop();
                timeInputLayer += timer.Elapsed();

                // Forward pass: Hidden layer 1
                timer.Start();
                dim3 hiddenGridDim((HIDDEN_SIZE_1 + blockDim.x - 1) / blockDim.x);
                forwardLayerGPU<<<hiddenGridDim, blockDim>>>(d_inputLayer, d_hiddenWeights1, d_hiddenBiases1, d_hiddenLayer1, INPUT_SIZE, HIDDEN_SIZE_1, true);
                hipDeviceSynchronize();
                timer.Stop();
                timeHiddenLayer1 += timer.Elapsed();

                // Forward pass: Hidden layer 2
                timer.Start();
                forwardLayerGPU<<<hiddenGridDim, blockDim>>>(d_hiddenLayer1, d_hiddenWeights2, d_hiddenBiases2, d_hiddenLayer2, HIDDEN_SIZE_1, HIDDEN_SIZE_2, true);
                hipDeviceSynchronize();
                timer.Stop();
                timeHiddenLayer2 += timer.Elapsed();

                // Forward pass: Output layer
                timer.Start();
                dim3 outputGridDim((OUTPUT_SIZE + blockDim.x - 1) / blockDim.x);
                forwardLayerGPU<<<outputGridDim, blockDim>>>(d_hiddenLayer2, d_outputWeights, d_outputBiases, d_outputLayer, HIDDEN_SIZE_2, OUTPUT_SIZE, false);
                softmaxGPU<<<1, OUTPUT_SIZE>>>(d_outputLayer, OUTPUT_SIZE);
                hipDeviceSynchronize();
                timer.Stop();
                timeOutputLayer += timer.Elapsed();

                // Compute output delta
                calculateOutputDeltaGPU<<<outputGridDim, blockDim>>>(d_outputDelta, d_outputLayer, d_trainLabels, OUTPUT_SIZE, 1);
                hipDeviceSynchronize();

                // Backpropagation: Update weights for output layer
                timer.Start();
                updateWeightsGPU<<<outputGridDim, blockDim>>>(d_outputWeights, d_outputBiases, d_hiddenLayer2, d_outputDelta, OUTPUT_SIZE, HIDDEN_SIZE_2, LEARNING_RATE);
                hipDeviceSynchronize();
                timer.Stop();
                timeHidden2Output += timer.Elapsed();

                // Backpropagation: Compute delta for hidden layer 2
                timer.Start();
                computeDeltaGPU<<<hiddenGridDim, blockDim>>>(d_hiddenLayer2, d_outputDelta, d_hiddenLayer2Delta, d_outputWeights, HIDDEN_SIZE_2, OUTPUT_SIZE);
                hipDeviceSynchronize();

                // Update weights for hidden layer 2
                updateWeightsGPU<<<hiddenGridDim, blockDim>>>(d_hiddenWeights2, d_hiddenBiases2, d_hiddenLayer1, d_hiddenLayer2Delta, HIDDEN_SIZE_2, HIDDEN_SIZE_1, LEARNING_RATE);
                hipDeviceSynchronize();
                timer.Stop();
                timeHidden1Hidden2 += timer.Elapsed();

                // Backpropagation: Compute delta for hidden layer 1
                timer.Start();
                computeDeltaGPU<<<hiddenGridDim, blockDim>>>(d_hiddenLayer1, d_hiddenLayer2Delta, d_hiddenLayer1Delta, d_hiddenWeights2, HIDDEN_SIZE_1, HIDDEN_SIZE_2);
                hipDeviceSynchronize();

                // Update weights for hidden layer 1
                updateWeightsGPU<<<hiddenGridDim, blockDim>>>(d_hiddenWeights1, d_hiddenBiases1, d_inputLayer, d_hiddenLayer1Delta, HIDDEN_SIZE_1, INPUT_SIZE, LEARNING_RATE);
                hipDeviceSynchronize();
                timer.Stop();
                timeInputHidden1 += timer.Elapsed();
            }
            if (epoch % 10 == 0) {  // Print accuracy every 10 epochs
                float accuracy = computeAccuracyGPU(d_hiddenWeights1, d_hiddenWeights2, d_outputWeights,
                                                    d_hiddenBiases1, d_hiddenBiases2, d_outputBiases,
                                                    testImages, testLabels, numTestImages);
                printf("Epoch %d: Accuracy = %.2f%%\n", epoch, accuracy * 100);
            }        
        }
        finalAccuracy = computeAccuracyGPU(d_hiddenWeights1, d_hiddenWeights2, d_outputWeights, d_hiddenBiases1, d_hiddenBiases2, d_outputBiases, testImages, testLabels, numTestImages);
        // Free device memory
        hipFree(d_inputLayer);
        hipFree(d_hiddenWeights1);
        hipFree(d_hiddenWeights2);
        hipFree(d_outputWeights);
        hipFree(d_hiddenBiases1);
        hipFree(d_hiddenBiases2);
        hipFree(d_outputBiases);
        hipFree(d_hiddenLayer1);
        hipFree(d_hiddenLayer2);
        hipFree(d_outputLayer);
        hipFree(d_outputDelta);
        hipFree(d_hiddenLayer2Delta);
        hipFree(d_hiddenLayer1Delta);
        hipFree(d_trainLabels);

    } 
    else {
        // Xử lý với kernel2
        for (int i = 1; i <= EPOCHS; i++)
        {
            //TODO
            // Xử lý lớp input
            timer.Start();
            //....
            timer.Stop();
            timeInputLayer += timer.Elapsed();

            // Xử lý lớp hidden 2
            timer.Start();
            //....
            timer.Stop();
            timeHiddenLayer1 += timer.Elapsed();

            // Xử lý lớp hidden 1
            timer.Start();
            //....
            timer.Stop();
            timeHiddenLayer2 += timer.Elapsed();

            // Xử lý lớp output
            timer.Start();
            //....
            timer.Stop();
            timeOutputLayer += timer.Elapsed();
            
            // Tính toán acc và loss
            //....
       
        }
    }
    printf("\nQuá trình feedforward:\n");
    printf("Thời gian chạy trung bình ở lớp Input trong 1 epoch là: %f\n", timeInputLayer / EPOCHS);
    printf("Thời gian chạy trung bình ở lớp Hidden 1 trong 1 epoch là: %f\n", timeHiddenLayer1 / EPOCHS);
    printf("Thời gian chạy trung bình ở lớp Hidden 2 trong 1 epoch là: %f\n", timeHiddenLayer2 / EPOCHS);
    printf("Thời gian chạy trung bình ở lớp Output trong 1 epoch là: %f\n", timeOutputLayer / EPOCHS);

    printf("\nQuá trình backpropagation:\n");
    printf("Thời gian cập nhật trọng số trung bình từ hidden 1 về input trong 1 epoch là: %f\n", timeInputHidden1 / EPOCHS);
    printf("Thời gian cập nhật trọng số trung bình từ hidden 2 về hidden 1 trong 1 epoch là: %f\n", timeHidden1Hidden2 / EPOCHS);
    printf("Thời gian cập nhật trọng số trung bình từ output về hidden 2 trong 1 epoch là: %f\n", timeHidden2Output / EPOCHS);

    printf("Độ chính xác của mô hình trên tập test là: %.2f%%\n", finalAccuracy * 100);

    return 0;
}

int main() {
    const char* trainImageFile = "./dataset/train/train-images-idx3-ubyte";
    const char* trainLabelFile = "./dataset/train/train-labels-idx1-ubyte";
    const char* testImageFile = "./dataset/test/t10k-images-idx3-ubyte";
    const char* testLabelFile = "./dataset/test/t10k-labels-idx1-ubyte";

    int numTrainImages, numtestImages, numRows, numCols;
    int numTrainLabels, numtestLabels;

    // Đọc tập tin ảnh
    unsigned char** trainImages = readImages(trainImageFile, &numTrainImages, &numRows, &numCols);
    unsigned char** testImages = readImages(testImageFile, &numtestImages, &numRows, &numCols);

    if (!trainImages || !testImages) {
        printf("Error reading images file.\n");
        return 1;
    }

    // Đọc tập tin nhãn
    unsigned char* trainLabels = readLabels(trainLabelFile, &numTrainLabels);
    unsigned char* testLabels = readLabels(testLabelFile, &numtestLabels);

    if (!trainLabels || !testLabels) {
        printf("Error reading labels file.\n");
        return 1;
    }

    // Hiển thị thông tin tập huấn luyện
    printf("Training Set:\n");
    printf("Number of training images: %d\n", numTrainImages);
    printf("Training image size: %dx%d\n", numRows, numCols);
    printf("Number of training labels: %d\n", numTrainLabels);

    // Hiển thị thông tin tập kiểm tra
    printf("Test Set:\n");
    printf("Number of Test images: %d\n", numtestImages);
    printf("Test image size: %dx%d\n", numRows, numCols);
    printf("Number of Test labels: %d\n\n", numtestLabels);

    // Khởi chạy với host
    printf("### Huấn luyện mô hình với Host ###\n\n");
    training(trainImages, trainLabels, testImages, testLabels, numTrainImages, numtestImages, numRows, numCols, 1);

    // Khởi chạy với kernel1
    printf("\n### Huấn luyện mô hình với kernel1 ###\n\n");
    training(trainImages, trainLabels, testImages, testLabels, numTrainImages, numtestImages, numRows, numCols, 2);

    // Khởi chạy với kernel2
    printf("\n### Huấn luyện mô hình với kernel2 ###\n\n");
    training(trainImages, trainLabels, testImages, testLabels, numTrainImages, numtestImages, numRows, numCols, 3);

    // Giải phóng bộ nhớ
    for (int i = 0; i < numTrainImages; i++) {
        free(trainImages[i]);
    }
    free(trainImages);

    for (int i = 0; i < numtestImages; i++) {
        free(testImages[i]);
    }
    free(testImages);

    free(trainLabels);
    free(testLabels);

    return 0;
}
