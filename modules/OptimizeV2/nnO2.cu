#include "hip/hip_runtime.h"
#include "nnO2.h"

__host__ __device__ float sigmoidKernel1(float x)
{
    return 1.0f / (1.0f + expf(-x));
}

__host__ __device__ float sigmoidDerivativeKernel1(float x)
{
    return x * (1.0f - x);
}

__global__ void softmaxKernel1(float *x, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= size)
        return;

    float maxVal = -FLT_MAX;
    for (int i = 0; i < size; i++)
        maxVal = max(maxVal, x[i]);

    float expSum = 0.0f;
    for (int i = 0; i < size; i++)
        expSum += expf(x[i] - maxVal);

    x[tid] = expf(x[tid] - maxVal) / expSum;
}

__global__ void forwardLayerKernel3(float *inputLayer, float *weights, float *biases, float *outputLayer, int inputSize, int outputSize, bool applySigmoid) {
    extern __shared__ float sharedResult[];
    if (threadIdx.x < inputSize) {
        sharedResult[threadIdx.x] = inputLayer[threadIdx.x] * weights[inputSize * blockIdx.x + threadIdx.x]; 
    }
    __syncthreads();

    for (int stride = (blockDim.x - 1) / 2 + 1; stride > 0; )
    {
        if (threadIdx.x < stride && (threadIdx.x + stride) < blockDim.x) {
            sharedResult[threadIdx.x] += sharedResult[threadIdx.x + stride];
        }
        __syncthreads();   

        if (stride == 1)
            break;
        stride = (stride - 1) / 2 + 1;
    }

    if (threadIdx.x == 0)
    {
        if (applySigmoid)
            sharedResult[0] = sigmoidKernel1(sharedResult[0] + biases[blockIdx.x]);
        outputLayer[blockIdx.x] = sharedResult[0];
    }
}

__global__ void calculateCValueKernel1(float *outputDelta, float *outputLayer, unsigned char *trainLabels, int i)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    outputDelta[j] = (trainLabels[i] == j ? 1.0f : 0.0f) - outputLayer[j];
}

__global__ void calculateDeltaLayerKernel3(float *currentLayer, float *nextLayerDelta, float *currentLayerDelta, float *weights, int currentLayerSize, int nextLayerSize)
{
    extern __shared__ float sharedResult[];
    if (threadIdx.x < nextLayerSize)
        sharedResult[threadIdx.x] = nextLayerDelta[threadIdx.x]  * weights[blockIdx.x * nextLayerSize + threadIdx.x];
    __syncthreads();

    for (int stride = (blockDim.x - 1) / 2 + 1; stride > 0; )
    {
        if (threadIdx.x < stride && (threadIdx.x + stride) < blockDim.x)
            sharedResult[threadIdx.x] += sharedResult[threadIdx.x + stride];
        __syncthreads();   

        if (stride == 1)
            break;

        stride = (stride - 1) / 2 + 1;
    }

    if (threadIdx.x == 0)
        currentLayerDelta[blockIdx.x] = sharedResult[0] * sigmoidDerivativeKernel1(currentLayer[blockIdx.x]);
}

__global__ void updateWeightsKernel1(float *weights, const float *layer, const float *delta, int layerSize, int prevLayerSize, float learningRate)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < layerSize)
        for (int k = 0; k < prevLayerSize; k++)
            weights[k * layerSize + i] += learningRate * layer[k] * delta[i];
}

__global__ void updateBiasesKernel1(float *biases, const float *delta, int layerSize, float learningRate)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < layerSize)
        biases[j] += learningRate * delta[j];
}

__global__ void createInputLayerKernel1(unsigned char *image, int inputSize, float *inputLayer)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < inputSize)
        inputLayer[i] = image[i] / 255.0f;
}

returnStruct trainKernel3(unsigned char **trainImages, unsigned char *trainLabels, unsigned char **testImages, unsigned char *testLabels, int numTrainImages, int numTestImages, int numRows, int numCols, int numEpochs) {
    GpuTimer timer;
    float timeInputLayer = 0, timeHiddenLayer1 = 0, timeHiddenLayer2 = 0, timeOutputLayer = 0;
    float timeInputHidden1 = 0, timeHidden1Hidden2 = 0, timeHidden2Output = 0;
    float finalAccuracy = 0;
    returnStruct result;

    // Initialize
    float *d_hiddenWeights1, *d_hiddenWeights2, *d_outputWeights;
    float *d_hiddenBiases1, *d_hiddenBiases2, *d_outputBiases;
    unsigned char *d_trainImages, *d_testImages, *d_trainLabels, *d_testLabels;

    // Allocate memory on GPU
    CHECK(hipMalloc((void **)&d_hiddenWeights1, INPUT_SIZE * HIDDEN_SIZE_1 * sizeof(float)));
    CHECK(hipMalloc((void **)&d_hiddenWeights2, HIDDEN_SIZE_1 * HIDDEN_SIZE_2 * sizeof(float)));
    CHECK(hipMalloc((void **)&d_outputWeights, HIDDEN_SIZE_2 * OUTPUT_SIZE * sizeof(float)));
    CHECK(hipMalloc((void **)&d_hiddenBiases1, HIDDEN_SIZE_1 * sizeof(float)));
    CHECK(hipMalloc((void **)&d_hiddenBiases2, HIDDEN_SIZE_2 * sizeof(float)));
    CHECK(hipMalloc((void **)&d_outputBiases, OUTPUT_SIZE * sizeof(float)));
    CHECK(hipMalloc((void **)&d_trainImages, numTrainImages * INPUT_SIZE * sizeof(unsigned char)));
    CHECK(hipMalloc((void **)&d_trainLabels, numTrainImages * sizeof(unsigned char)));
    CHECK(hipMalloc((void **)&d_testImages, numTestImages * INPUT_SIZE * sizeof(unsigned char)));
    CHECK(hipMalloc((void **)&d_testLabels, numTestImages * sizeof(unsigned char)));
    float *h_hiddenWeights1 = (float *)malloc(INPUT_SIZE * HIDDEN_SIZE_1 * sizeof(float));
    float *h_hiddenWeights2 = (float *)malloc(HIDDEN_SIZE_1 * HIDDEN_SIZE_2 * sizeof(float));
    float *h_outputWeights = (float *)malloc(HIDDEN_SIZE_2 * OUTPUT_SIZE * sizeof(float));
    float *h_hiddenBiases1 = (float *)malloc(HIDDEN_SIZE_1 * sizeof(float));
    float *h_hiddenBiases2 = (float *)malloc(HIDDEN_SIZE_2 * sizeof(float));
    float *h_outputBiases = (float *)malloc(OUTPUT_SIZE * sizeof(float));

    // Initialize weights and biases
    for (int i = 0; i < INPUT_SIZE * HIDDEN_SIZE_1; i++)
        h_hiddenWeights1[i] = ((float)rand() / RAND_MAX - 0.5f) * 2;
    for (int i = 0; i < HIDDEN_SIZE_1 * HIDDEN_SIZE_2; i++)
        h_hiddenWeights2[i] = ((float)rand() / RAND_MAX - 0.5f) * 2;
    for (int i = 0; i < HIDDEN_SIZE_2 * OUTPUT_SIZE; i++)
        h_outputWeights[i] = ((float)rand() / RAND_MAX - 0.5f) * 2;
    for (int i = 0; i < HIDDEN_SIZE_1; i++)
        h_hiddenBiases1[i] = 0;
    for (int i = 0; i < HIDDEN_SIZE_2; i++)
        h_hiddenBiases2[i] = 0;
    for (int i = 0; i < OUTPUT_SIZE; i++)
        h_outputBiases[i] = 0;

    // Copy data to device
    CHECK(hipMemcpy(d_hiddenWeights1, h_hiddenWeights1, INPUT_SIZE * HIDDEN_SIZE_1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_hiddenWeights2, h_hiddenWeights2, HIDDEN_SIZE_1 * HIDDEN_SIZE_2 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_outputWeights, h_outputWeights, HIDDEN_SIZE_2 * OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_hiddenBiases1, h_hiddenBiases1, HIDDEN_SIZE_1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_hiddenBiases2, h_hiddenBiases2, HIDDEN_SIZE_2 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_outputBiases, h_outputBiases, OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_trainLabels, trainLabels, numTrainImages * sizeof(unsigned char), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_testLabels, testLabels, numTestImages * sizeof(unsigned char), hipMemcpyHostToDevice));

    // Flatten images
    unsigned char *h_trainImages = (unsigned char *)malloc(numTrainImages * INPUT_SIZE * sizeof(unsigned char));
    unsigned char *h_testImages = (unsigned char *)malloc(numTestImages * INPUT_SIZE * sizeof(unsigned char));

    for (int i = 0; i < numTrainImages; i++)
        for (int j = 0; j < INPUT_SIZE; j++)
            h_trainImages[i * INPUT_SIZE + j] = trainImages[i][j];

    for (int i = 0; i < numTestImages; i++)
        for (int j = 0; j < INPUT_SIZE; j++)
            h_testImages[i * INPUT_SIZE + j] = testImages[i][j];

    CHECK(hipMemcpy(d_trainImages, h_trainImages, numTrainImages * INPUT_SIZE * sizeof(unsigned char), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_testImages, h_testImages, numTestImages * INPUT_SIZE * sizeof(unsigned char), hipMemcpyHostToDevice));

    dim3 blockSize;
    dim3 gridSize;
    int sharedMemorySize;

    // Allocate memory for layers
    float *d_inputLayer, *d_hiddenLayer1, *d_hiddenLayer2, *d_outputLayer, *d_outputDelta, *d_hiddenLayer2Delta, *d_hiddenLayer1Delta;

    CHECK(hipMalloc((void **)&d_inputLayer, INPUT_SIZE * sizeof(float)));
    CHECK(hipMalloc((void **)&d_hiddenLayer1, HIDDEN_SIZE_1 * sizeof(float)));
    CHECK(hipMalloc((void **)&d_hiddenLayer2, HIDDEN_SIZE_2 * sizeof(float)));
    CHECK(hipMalloc((void **)&d_outputLayer, OUTPUT_SIZE * sizeof(float)));
    CHECK(hipMalloc((void **)&d_outputDelta, OUTPUT_SIZE * sizeof(float)));
    CHECK(hipMalloc((void **)&d_hiddenLayer2Delta, HIDDEN_SIZE_2 * sizeof(float)));
    CHECK(hipMalloc((void **)&d_hiddenLayer1Delta, HIDDEN_SIZE_1 * sizeof(float)));

    // Training
    for (int epoch = 1; epoch <= numEpochs; epoch++)
    {
        for (int batchStart = 0; batchStart < numTrainImages; batchStart += BATCH_SIZE)
        {
            int batchSize = fmin(BATCH_SIZE, numTrainImages - batchStart);
            for (int i = batchStart; i < batchStart + batchSize; i++)
            {
                // Forward pass
                // Input
                timer.Start();
                blockSize = dim3(128);
                gridSize = ((INPUT_SIZE - 1) / blockSize.x + 1);
                createInputLayerKernel1<<<gridSize, blockSize>>>(d_trainImages + i * INPUT_SIZE, INPUT_SIZE, d_inputLayer);
                timer.Stop();
                timeInputLayer += timer.Elapsed();

                // Hidden 1
                timer.Start();
                blockSize = dim3(INPUT_SIZE);
                gridSize = ((HIDDEN_SIZE_1 * INPUT_SIZE - 1) / blockSize.x + 1);
                sharedMemorySize = sizeof(float) * INPUT_SIZE;
                forwardLayerKernel3<<<gridSize, blockSize, sharedMemorySize>>>(d_inputLayer, d_hiddenWeights1, d_hiddenBiases1, d_hiddenLayer1, INPUT_SIZE, HIDDEN_SIZE_1, true);
                timer.Stop();
                timeHiddenLayer1 += timer.Elapsed();

                // Hidden 2
                timer.Start();
                blockSize = dim3(HIDDEN_SIZE_1);
                gridSize = ((HIDDEN_SIZE_2 * HIDDEN_SIZE_1 - 1) / blockSize.x + 1);
                sharedMemorySize = sizeof(float) * HIDDEN_SIZE_1;
                forwardLayerKernel3<<<gridSize, blockSize, sharedMemorySize>>>(d_hiddenLayer1, d_hiddenWeights2, d_hiddenBiases2, d_hiddenLayer2, HIDDEN_SIZE_1, HIDDEN_SIZE_2, true);
                timer.Stop();
                timeHiddenLayer2 += timer.Elapsed();

                // Output
                timer.Start();
                blockSize = dim3(HIDDEN_SIZE_2);
                gridSize = ((OUTPUT_SIZE * HIDDEN_SIZE_2 - 1) / blockSize.x + 1);
                sharedMemorySize = sizeof(float) * HIDDEN_SIZE_2;
                forwardLayerKernel3<<<gridSize, blockSize, sharedMemorySize>>>(d_hiddenLayer2, d_outputWeights, d_outputBiases, d_outputLayer, HIDDEN_SIZE_2, OUTPUT_SIZE, false);

                // Softmax
                blockSize = dim3(OUTPUT_SIZE);
                gridSize = ((OUTPUT_SIZE - 1) / blockSize.x + 1);
                softmaxKernel1<<<gridSize, blockSize>>>(d_outputLayer, OUTPUT_SIZE);
                timer.Stop();
                timeOutputLayer += timer.Elapsed();

                // Backpropagation
                // Backpropagation and update weights and biases
                timer.Start();
                blockSize = dim3(1);
                gridSize = ((OUTPUT_SIZE - 1) / blockSize.x + 1);
                calculateCValueKernel1<<<gridSize, blockSize>>>(d_outputDelta, d_outputLayer, d_trainLabels, i);

                // Gradient and update weights for output layer
                blockSize = dim3(HIDDEN_SIZE_2);
                gridSize = ((OUTPUT_SIZE * HIDDEN_SIZE_2 - 1) / blockSize.x + 1);
                sharedMemorySize = sizeof(float) * HIDDEN_SIZE_2;
                updateWeightsKernel2<<<gridSize, blockSize, sharedMemorySize>>>(d_outputWeights, d_hiddenLayer2, d_outputDelta, OUTPUT_SIZE, HIDDEN_SIZE_2);
                blockSize = dim3(128);
                gridSize = ((OUTPUT_SIZE - 1) / blockSize.x + 1);
                updateBiasesKernel1<<<gridSize, blockSize>>>(d_outputBiases, d_outputDelta, OUTPUT_SIZE, LEARNING_RATE);
                timer.Stop();
                timeHidden2Output += timer.Elapsed();

                // Gradient and update weights for hidden layer 2
                timer.Start();
                blockSize = dim3(OUTPUT_SIZE);
                gridSize = ((HIDDEN_SIZE_2 * OUTPUT_SIZE - 1) / blockSize.x + 1);
                sharedMemorySize = sizeof(float) * OUTPUT_SIZE;
                calculateDeltaLayerKernel3<<<gridSize, blockSize, sharedMemorySize>>>(d_hiddenLayer2, d_outputDelta, d_hiddenLayer2Delta, d_outputWeights, HIDDEN_SIZE_2, OUTPUT_SIZE);

                blockSize = dim3(HIDDEN_SIZE_1);
                gridSize = ((HIDDEN_SIZE_1 * HIDDEN_SIZE_2 - 1) / blockSize.x + 1);
                sharedMemorySize = sizeof(float) * HIDDEN_SIZE_1;
                updateWeightsKernel2<<<gridSize, blockSize, sharedMemorySize>>>(d_hiddenWeights2, d_hiddenLayer1, d_hiddenLayer2Delta, HIDDEN_SIZE_2, HIDDEN_SIZE_1);

                blockSize = dim3(128);
                gridSize = ((HIDDEN_SIZE_2 - 1) / blockSize.x + 1);
                updateBiasesKernel1<<<gridSize, blockSize>>>(d_hiddenBiases2, d_hiddenLayer2Delta, HIDDEN_SIZE_2, LEARNING_RATE);
                timer.Stop();
                timeHidden1Hidden2 += timer.Elapsed();

                // Gradient and update weights for hidden layer 1
                timer.Start();
                blockSize = dim3(HIDDEN_SIZE_2);
                gridSize = ((HIDDEN_SIZE_1 * HIDDEN_SIZE_2 - 1) / blockSize.x + 1);
                sharedMemorySize = sizeof(float) * HIDDEN_SIZE_2;
                calculateDeltaLayerKernel3<<<gridSize, blockSize, sharedMemorySize>>>(d_hiddenLayer1, d_hiddenLayer2Delta, d_hiddenLayer1Delta, d_hiddenWeights2, HIDDEN_SIZE_1, HIDDEN_SIZE_2);

                blockSize = dim3(INPUT_SIZE);
                gridSize = ((INPUT_SIZE * HIDDEN_SIZE_1 - 1) / blockSize.x + 1);
                sharedMemorySize = sizeof(float) * INPUT_SIZE;
                updateWeightsKernel2<<<gridSize, blockSize, sharedMemorySize>>>(d_hiddenWeights1, d_inputLayer, d_hiddenLayer1Delta, HIDDEN_SIZE_1, INPUT_SIZE);

                blockSize = dim3(128);
                gridSize = ((HIDDEN_SIZE_1 - 1) / blockSize.x + 1);
                updateBiasesKernel1<<<gridSize, blockSize>>>(d_hiddenBiases1, d_hiddenLayer1Delta, HIDDEN_SIZE_1, LEARNING_RATE);
                timer.Stop();
                timeInputHidden1 += timer.Elapsed();
            }
        }

        hipDeviceSynchronize();
        CHECK(hipGetLastError());

        // Copy result from device memory
        CHECK(hipMemcpy(h_hiddenWeights1, d_hiddenWeights1, INPUT_SIZE * HIDDEN_SIZE_1 * sizeof(float), hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(h_hiddenWeights2, d_hiddenWeights2, HIDDEN_SIZE_1 * HIDDEN_SIZE_2 * sizeof(float), hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(h_outputWeights, d_outputWeights, HIDDEN_SIZE_2 * OUTPUT_SIZE * sizeof(float), hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(h_hiddenBiases1, d_hiddenBiases1, HIDDEN_SIZE_1 * sizeof(float), hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(h_hiddenBiases2, d_hiddenBiases2, HIDDEN_SIZE_2 * sizeof(float), hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(h_outputBiases, d_outputBiases, OUTPUT_SIZE * sizeof(float), hipMemcpyDeviceToHost));

        printf("Epoch %d: Accuracy = %.4f\n", epoch, computeFinalAccuracy(testImages, testLabels, numTestImages, numRows, numCols, h_hiddenWeights1, h_hiddenWeights2, h_outputWeights, h_hiddenBiases1, h_hiddenBiases2, h_outputBiases, BATCH_SIZE));
    }
    finalAccuracy = computeFinalAccuracy(testImages, testLabels, numTestImages, numRows, numCols, h_hiddenWeights1, h_hiddenWeights2, h_outputWeights, h_hiddenBiases1, h_hiddenBiases2, h_outputBiases, BATCH_SIZE);

    free(h_hiddenWeights1);
    free(h_hiddenWeights2);
    free(h_outputWeights);
    free(h_hiddenBiases1);
    free(h_hiddenBiases2);
    free(h_outputBiases);
    free(h_trainImages);
    free(h_testImages);

    CHECK(hipFree(d_hiddenWeights1));
    CHECK(hipFree(d_hiddenWeights2));
    CHECK(hipFree(d_outputWeights));
    CHECK(hipFree(d_hiddenBiases1));
    CHECK(hipFree(d_hiddenBiases2));
    CHECK(hipFree(d_outputBiases));
    CHECK(hipFree(d_trainImages));
    CHECK(hipFree(d_trainLabels));
    CHECK(hipFree(d_testImages));
    CHECK(hipFree(d_testLabels));
    CHECK(hipFree(d_inputLayer));
    CHECK(hipFree(d_hiddenLayer1));
    CHECK(hipFree(d_hiddenLayer2));
    CHECK(hipFree(d_outputLayer));
    CHECK(hipFree(d_outputDelta));
    CHECK(hipFree(d_hiddenLayer2Delta));
    CHECK(hipFree(d_hiddenLayer1Delta));

    result.timeInputLayer = timeInputLayer;
    result.timeHiddenLayer1 = timeHiddenLayer1;
    result.timeHiddenLayer2 = timeHiddenLayer2;
    result.timeOutputLayer = timeOutputLayer;
    result.timeInputHidden1 = timeInputHidden1;
    result.timeHidden1Hidden2 = timeHidden1Hidden2;
    result.timeHidden2Output = timeHidden2Output;
    result.finalAccuracy = finalAccuracy;

    return result;
}


float computeFinalAccuracy(unsigned char **testImages, unsigned char *testLabels, int numTestImages, int numRows, int numCols, float *hiddenWeights1, float *hiddenWeights2, float *outputWeights, float *hiddenBiases1, float *hiddenBiases2, float *outputBiases, int batchSize)
{
    int correct = 0;
    for (int i = 0; i < numTestImages; i++)
    {
        // Feedforward
        float inputLayer[INPUT_SIZE];
        for (int j = 0; j < INPUT_SIZE; j++)
            inputLayer[j] = testImages[i][j] / 255.0f;

        // Lớp ẩn 1
        float *hiddenLayer1 = (float *)malloc(HIDDEN_SIZE_1 * sizeof(float));
        forwardLayer(inputLayer, hiddenWeights1, hiddenBiases1, hiddenLayer1, INPUT_SIZE, HIDDEN_SIZE_1);

        float *hiddenLayer2 = (float *)malloc(HIDDEN_SIZE_2 * sizeof(float));
        forwardLayer(hiddenLayer1, hiddenWeights2, hiddenBiases2, hiddenLayer2, HIDDEN_SIZE_1, HIDDEN_SIZE_2);

        float *outputLayer = (float *)malloc(OUTPUT_SIZE * sizeof(float));
        forwardLayer(hiddenLayer2, outputWeights, outputBiases, outputLayer, HIDDEN_SIZE_2, OUTPUT_SIZE, false);

        // Softmax
        softmax(outputLayer, OUTPUT_SIZE);

        // Check predict
        int predictedLabel = 0;
        for (int j = 1; j < OUTPUT_SIZE; j++)
            if (outputLayer[j] > outputLayer[predictedLabel])
                predictedLabel = j;

        if (predictedLabel == testLabels[i])
            correct++;
    }
    return (float)correct / numTestImages;
}
